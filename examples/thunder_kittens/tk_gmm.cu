#include "hip/hip_runtime.h"
#include <hip/hip_bf16.h>

#include <cstdlib>

#include "kittens.cuh"
#include "prototype.cuh"

namespace examples::thunder_kittens {

float *to_device(const std::vector<float> &vec) {
  float *d_ptr;
  hipMalloc(&d_ptr, vec.size() * sizeof(float));
  hipMemcpy(d_ptr, vec.data(), vec.size() * sizeof(float),
             hipMemcpyHostToDevice);
  return d_ptr;
}

template <int M_BLOCK, int N_BLOCK>
struct matmul_layout {
  using base_tile = kittens::st_bf<64, 64>;
  using global_layout = kittens::gl<kittens::bf16, 1, 1, -1, -1, base_tile>;
  struct globals {
    global_layout A, B, C;
  };
  struct input_block {
    base_tile a[M_BLOCK], b[N_BLOCK];
  };
  struct finish_block {
    base_tile c[M_BLOCK][N_BLOCK];
  };
  struct common_state {
    int2 coord;
  };
  struct consumer_state {
    kittens::rt_fl<16, N_BLOCK * base_tile::cols> accum;
  };
};

template <int _M_BLOCK = 2, int _N_BLOCK = 4, int _SUPER_M = 12>
struct matmul_template {
  static constexpr int M_BLOCK = _M_BLOCK, N_BLOCK = _N_BLOCK,
                       SUPER_M = _SUPER_M;
  using layout = matmul_layout<M_BLOCK, N_BLOCK>;
  using wide_tile = kittens::st_bf<64, 64 * N_BLOCK>;
  static constexpr int NUM_CONSUMER_WARPS = M_BLOCK * 4, INPUT_PIPE_STAGES = 4,
                       PRODUCER_BARRIER_ARRIVALS = 1;
  // Helper functions
  template <bool PERISISTENT_GRID = true>
  __host__ static inline dim3 grid(int M, int N, int K) {
    return dim3(
        PERISISTENT_GRID
            ? 132
            : M * N / (M_BLOCK * N_BLOCK * layout::base_tile::num_elements));
  }
  // ThunderKittens template functions
  __device__ static inline void common_setup(
      kittens::prototype::lcf::common_setup_args<layout> args) {
    int Rblocks = args.globals.C.rows() / (M_BLOCK * 64),
        Cblocks = args.globals.C.cols() / (N_BLOCK * 64);
    int super_rows = (Rblocks / SUPER_M) * SUPER_M,
        final_rows = Rblocks - super_rows, super_repeat = SUPER_M * Cblocks;
    int task_id = args.task_iter * gridDim.x + blockIdx.x;
    if (task_id < super_rows * Cblocks)
      args.common.coord = {
          SUPER_M * (task_id / super_repeat) + task_id % SUPER_M,
          (task_id % super_repeat) / SUPER_M};
    else if (task_id < Rblocks * Cblocks) {
      int remainder_id = task_id - super_rows * Cblocks;
      args.common.coord = {super_rows + (remainder_id % final_rows),
                           remainder_id / final_rows};
    } else {  // Id is too high, no more work to do
      args.num_iters = -1;
      return;
    }
    args.num_iters = args.globals.A.cols() / 64;
    int id = kittens::warpgroup::groupid() == NUM_CONSUMER_WARPS / 4
                 ? 0
                 : kittens::warpgroup::groupid();  // producer sets as 0
    args.common.coord = {args.common.coord.x * M_BLOCK + id,
                         args.common.coord.y * N_BLOCK};
  }
  struct producer {
    __device__ static void setup(
        kittens::prototype::lcf::producer_setup_args<layout> args) {
      kittens::warpgroup::decrease_registers<40>();  // decrease registers for
                                                     // producers
    }
    __device__ static void load(
        kittens::prototype::lcf::producer_load_args<layout> args) {
      if (kittens::warpgroup::warpid() == 0) {
        kittens::tma::expect(args.inputs_arrived, args.input);
        for (int i = 0; i < M_BLOCK; i++)
          kittens::tma::load_async(args.input.a[i], args.globals.A,
                                   {args.common.coord.x + i, args.iter},
                                   args.inputs_arrived);
        for (int i = 0; i < N_BLOCK; i++)
          kittens::tma::load_async(args.input.b[i], args.globals.B,
                                   {args.iter, args.common.coord.y + i},
                                   args.inputs_arrived);
      }
    }
  };
  struct consumer {
    __device__ static void setup(
        kittens::prototype::lcf::consumer_setup_args<layout> args) {
      kittens::warpgroup::increase_registers<232>();  // increase registers for
                                                      // consumers
      zero(args.state.accum);
    }
    __device__ static void compute(
        kittens::prototype::lcf::consumer_compute_args<layout> args) {
      kittens::warpgroup::mma_AB(
          args.state.accum,                             // dest registers
          args.input.a[kittens::warpgroup::groupid()],  // A matrix
          reinterpret_cast<wide_tile &>(args.input.b)   // B matrix
      );
      kittens::warpgroup::mma_async_wait();
      if (kittens::laneid() == 0) kittens::arrive(args.inputs_finished);
    }
    __device__ static void finish(
        kittens::prototype::lcf::consumer_finish_args<layout> args) {
      kittens::warpgroup::store(
          reinterpret_cast<wide_tile &>(
              args.finish.c[kittens::warpgroup::groupid()]),
          args.state.accum);
      kittens::warpgroup::sync(kittens::warpgroup::groupid() + 4);
      if (kittens::warpgroup::warpid() == 0)
        for (int i = 0; i < N_BLOCK; i++) {
          kittens::tma::store_async(
              args.globals.C, args.finish.c[kittens::warpgroup::groupid()][i],
              {args.common.coord.x, args.common.coord.y + i});
          kittens::tma::store_async_read_wait();  // wait that store is finished
                                                  // before reusing finish
                                                  // memory
        }
      kittens::zero(args.state.accum);
      if (kittens::laneid() == 0) kittens::arrive(args.finish_finished);
    }
  };
};

template <typename mmt>
void inner_run(kittens::bf16 *d_A, kittens::bf16 *d_B, kittens::bf16 *d_C,
               size_t M, size_t N, size_t K, dim3 grid, dim3 block) {
  using global_layout = typename mmt::layout::global_layout;
  using globals = typename mmt::layout::globals;
  global_layout Ag{d_A, nullptr, nullptr, M, K};
  global_layout Bg{d_B, nullptr, nullptr, K, N};
  global_layout Cg{d_C, nullptr, nullptr, M, N};
  globals G{Ag, Bg, Cg};
  kittens::prototype::lcf::kernel<mmt>
      <<<grid, block, kittens::MAX_SHARED_MEMORY - 1024>>>(G);
}

void gmm_gpu(const float *A, const float *B, float alpha, float beta, int M,
             int K, int N, float *C) {
  // TODO: Tune.
  using mmt = matmul_template<2, 4, 8>;

  hipError_t cudaStatus;

  // Allocate device memory
  __hip_bfloat16 *d_A, *d_B, *d_C;
  hipMalloc(&d_A, M * K * sizeof(__hip_bfloat16));
  hipMalloc(&d_B, K * N * sizeof(__hip_bfloat16));
  hipMalloc(&d_C, M * N * sizeof(__hip_bfloat16));

  // Check for CUDA errors
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipError_t::hipSuccess) {
    std::cerr << "CUDA error: " << hipGetErrorString(cudaStatus) << std::endl;
    std::exit(EXIT_FAILURE);
  }

  // Convert to __hip_bfloat16 and copy to device
  __hip_bfloat16 *h_A_bf16 = new __hip_bfloat16[M * K];
  __hip_bfloat16 *h_B_bf16 = new __hip_bfloat16[K * N];
  std::transform(A, A + M * K, h_A_bf16,
                 [](float val) { return __hip_bfloat16(val); });
  std::transform(B, B + K * N, h_B_bf16,
                 [](float val) { return __hip_bfloat16(val); });
  hipMemcpy(d_A, h_A_bf16, M * K * 2, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B_bf16, K * N * 2, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  delete[] h_A_bf16;
  delete[] h_B_bf16;

  unsigned long mem_size = kittens::MAX_SHARED_MEMORY - 1024;
  hipFuncSetAttribute(reinterpret_cast<const void*>(kittens::prototype::lcf::kernel<mmt>),
                       hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);

  // Launch kernel
  dim3 grid(mmt::grid(M, N, K));
  dim3 block(kittens::prototype::detail::NUM_THREADS_v<mmt>);
  inner_run<mmt>(d_A, d_B, d_C, M, N, K, grid, block);
  hipDeviceSynchronize();

  // Check for CUDA errors
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipError_t::hipSuccess) {
    std::cerr << "CUDA error: " << hipGetErrorString(cudaStatus) << std::endl;
    std::exit(EXIT_FAILURE);
  }

  // Copy result back to host
  __hip_bfloat16 *h_C_bf16 = new __hip_bfloat16[M * N];
  hipMemcpy(h_C_bf16, d_C, M * N * 2, hipMemcpyDeviceToHost);

  // Clean up
  delete[] h_C_bf16;
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

}  // namespace examples::thunder_kittens
#include <gtest/gtest.h>

#include <iostream>
#include "examples/architectures/simple_kernel.cuh"

namespace examples::architectures {
    TEST(SimpleKernelTest, Increment) { 
        std::vector<float> data(1000, 0.0f);

        float* d_data;
        hipMalloc(&d_data, data.size() * sizeof(float));
        hipMemcpy(d_data, data.data(), data.size() * sizeof(float), hipMemcpyHostToDevice);

        increment_all(d_data, data.size());

        hipMemcpy(data.data(), d_data, data.size() * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(d_data);
        
        // for (const auto& value : data) {
        //     EXPECT_EQ(value, 1.0f);
        // }
    }
}  // namespace examples::architectures
